#include "hip/hip_runtime.h"
/*
 * Created on Tue Jun 23 2021
 *
 * Copyright (c) 2021 HITsz-NRSL
 *
 * Author: EpsAvlc
 */

#include "active_mapping/world_representation/cuda_acc/cuda_ray_caster.h"
// #include <octomap/octomap.h>
#include <limits>

namespace octomap {

/* const values */
const int kTreeMaxVal = 32768;

/* cuda utility functions */
__device__ CudaOcTreeKey coordToKey(double resolution_factor,
                                    octomap::CudaPoint3d coord) {
  CudaOcTreeKey key;
  for (int i = 0; i < 3; ++i) {
    key.k[i] = ((int)floor(resolution_factor * coord(i))) + kTreeMaxVal;
  }
  return key;
}

__device__ double keyToCoord(double resolution, uint16_t key) {
  return (double((int)key - (int)kTreeMaxVal) + 0.5) * resolution;
}

__device__ octomap::CudaPoint3d keyToCoord(double resolution,
                                           CudaOcTreeKey key) {
  octomap::CudaPoint3d pt;
  for (int i = 0; i < 3; ++i) {
    pt(i) = (float)keyToCoord(resolution, key.k[i]);
  }
  return pt;
}

__device__ cu_uint64_t keyToIndex(const CudaOcTreeKey &key) {
  cu_uint64_t res = 0;
  res = ((cu_uint64_t)key.k[0] << 32) + ((cu_uint64_t) key.k[1] << 16) + (key.k[2]);
  return res;
}

__global__ void
kernelCastRay(KeyValue *hash_table, double resolution,
              const unsigned int cast_num, octomap::CudaPoint3d *origins,
              octomap::CudaPoint3d *dirs, octomap::CudaPoint3d *end_pts,
              bool ignore_unknown, double *max_ranges, bool *find_end_pt) {
  // octomap::CudaOcTreeKey key;
  // key.k[0] = 32798;
  // key.k[1] = 32841;
  // key.k[2] = 32784;
  unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_id < cast_num) {
    double resolution_factor = 1 / resolution;
    // cu_uint64_t key_ind = keyToIndex(key);
    // cu_uint64_t key_occ = CudaHashTable::gpuQuery(hash_table, key_ind);
    // printf("key_ind: %lld, key_occ: %lld\n", key_ind, key_occ);
    octomap::CudaPoint3d origin = origins[thread_id];
    octomap::CudaPoint3d direction = dirs[thread_id].normalized();
    octomap::CudaPoint3d *end = &end_pts[thread_id];
    double max_range = max_ranges[thread_id];

    CudaOcTreeKey current_key = coordToKey(resolution_factor, origin);
    cu_uint64_t current_key_ind = keyToIndex(current_key);
    cu_uint64_t starting_node_occ =
        CudaHashTable::gpuQuery(hash_table, current_key_ind);
    if (starting_node_occ != vEmpty) {
      if (starting_node_occ == CudaRayCaster::OccStatus::Occupied) {
        *end = keyToCoord(resolution, current_key);
        find_end_pt[thread_id] = true;
        return;
      }
    } else if (!ignore_unknown) {
      *end = keyToCoord(resolution, current_key);
      find_end_pt[thread_id] = false;
      return;
    }

    bool max_range_set = (max_range > 0.0);

    int step[3];
    double tMax[3];
    double tDelta[3];

    for (unsigned int i = 0; i < 3; ++i) {
      // compute step direction
      if (direction(i) > 0.0)
        step[i] = 1;
      else if (direction(i) < 0.0)
        step[i] = -1;
      else
        step[i] = 0;

      // compute tMax, tDelta
      if (step[i] != 0) {
        // corner point of voxel (in direction of ray)
        double voxelBorder = keyToCoord(resolution, current_key.k[i]);
        voxelBorder += static_cast<double>(step[i] * resolution * 0.5);

        tMax[i] = (voxelBorder - origin(i)) / direction(i);
        tDelta[i] = resolution / fabs(direction(i));
      } else {
        tMax[i] = __DBL_MAX__;
        tDelta[i] = __DBL_MAX__;
      }
    }

    if (step[0] == 0 && step[1] == 0 && step[2] == 0) {
      printf(
          "[CudaRayCaster]:Raycasting in direction (0,0,0) is not possible!");
      find_end_pt[thread_id] = false;
      return;
    }
    // for speedup:
    float maxrange_sq = max_range * max_range;

    // Incremental phase
    // ---------------------------------------------------------
    bool done = false;

    while (!done) {
      // printf("[cuda_ray_caster] current key: %d, %d, %d \n", current_key.k[0], current_key.k[1], current_key.k[2]);
      unsigned int dim;

      // find minimum tMax:
      if (tMax[0] < tMax[1]) {
        if (tMax[0] < tMax[2])
          dim = 0;
        else
          dim = 2;
      } else {
        if (tMax[1] < tMax[2])
          dim = 1;
        else
          dim = 2;
      }

      // check for overflow:
      if ((step[dim] < 0 && current_key.k[dim] == 0) ||
          (step[dim] > 0 && current_key.k[dim] == 2 * kTreeMaxVal - 1)) {
        printf(
            "[CudaCastRay] Coordinate hit bounds in dim %d, aborting raycast\n",
            dim);
        // return border point nevertheless:
        *end = keyToCoord(resolution, current_key);
        find_end_pt[thread_id] = false;
        return;
      }

      // advance in direction "dim"
      current_key.k[dim] += step[dim];
      tMax[dim] += tDelta[dim];

      // generate world coords from key
      *end = keyToCoord(resolution, current_key);

      // check for maxrange:
      if (max_range_set) {
        // printf("max_range: %f\n", max_range);
        float dist_from_origin_sq = 0;
        for (unsigned int j = 0; j < 3; j++) {
          // printf("dist_from_origin_sq: %f\n", dist_from_origin_sq);
          float add_val = ((end->operator()(j) - origin(j)) *
          (end->operator()(j) - origin(j)));
          // printf("add %f\n", add_val);
          dist_from_origin_sq += add_val;
        }
        if (dist_from_origin_sq > maxrange_sq) {
          find_end_pt[thread_id] = false;
          // printf("[CudaRayCaster] reach max_range. max_range: %f, end_pt: %f, %f, %f, origin_pt: %f, %f, %f\n", 
          // max_range, end->operator()(0), end->operator()(1), end->operator()(2), origin(0), origin(1), origin(2));
          return;
        }
      }

      current_key_ind = keyToIndex(current_key);
      cu_uint64_t current_node_occ =
          CudaHashTable::gpuQuery(hash_table, current_key_ind);
      // printf("[cuda_ray_caster]current_key: %d, %d, %d, cuda_node_val : %lld\n", current_key.k[0], current_key.k[1], current_key.k[2], current_node_occ);
      if (current_node_occ != vEmpty) {
        if (current_node_occ == CudaRayCaster::OccStatus::Occupied) {
          done = true;
          // printf("[CudaRayCaster] hit occ in %d, %d, %d\n", current_key.k[0], current_key.k[1], current_key.k[2]);
          break;
        }
        // otherwise: node is free and valid, raycasting continues
      } else if (!ignore_unknown) {  // no node found, this usually means we are
                                     // in "unknown" areas
        find_end_pt[thread_id] = false;
        // printf("[CudaRayCaster] hit unknown in %d, %d, %d", current_key.k[0], current_key.k[1], current_key.k[2]);
        return;
      }
    }  // end while

    find_end_pt[thread_id] = true;
    return;
  }
}

/* CudaRayCaster methods. */
CudaRayCaster::CudaRayCaster(const OcTreeData &octree_data, bool print_info)
    : resolution_(octree_data.resolution) {
  int node_num = octree_data.keys.size();
  if (print_info) {
    std::cout << "[CudaRayCaster] octree node num: " << node_num << std::endl;
  }
  std::vector<KeyValue> kvs(node_num);
  for (int i = 0; i < node_num; ++i) {
    kvs[i].key = octree_data.keys[i];
    kvs[i].value = octree_data.occupancy[i];
  }
  cu_hash_table_.insert(kvs.data(), node_num);
}

bool*
CudaRayCaster::castRay(const std::vector<octomap::point3d> &origins,
                       const std::vector<octomap::point3d> &dirs,
                       std::vector<octomap::point3d> *end_pts,
                       bool ignore_unknown,
                       const std::vector<double> &max_range) {
  int cast_size = origins.size();
  CudaPoint3d *device_origins;
  hipMalloc(&device_origins, sizeof(point3d) * cast_size);
  hipMemcpy(device_origins, origins.data(), sizeof(point3d) * cast_size,
             hipMemcpyHostToDevice);

  CudaPoint3d *device_dirs;
  hipMalloc(&device_dirs, sizeof(point3d) * cast_size);
  hipMemcpy(device_dirs, dirs.data(), sizeof(point3d) * dirs.size(),
             hipMemcpyHostToDevice);

  CudaPoint3d *device_end_pts;
  hipMalloc(&device_end_pts, sizeof(point3d) * cast_size);

  double *device_max_ranges;
  hipMalloc(&device_max_ranges, sizeof(double) * cast_size);
  hipMemcpy(device_max_ranges, max_range.data(),
             sizeof(double) * cast_size, hipMemcpyHostToDevice);

  bool *device_find_end_pts;
  hipMalloc(&device_find_end_pts, sizeof(bool) * cast_size);

  int mingridsize;
  int threadblocksize;
  hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize,
                                     kernelCastRay, 0, 0);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  int gridsize =
      ((cu_uint64_t)origins.size() + threadblocksize - 1) / threadblocksize;
  kernelCastRay<<<gridsize, threadblocksize>>>(
      cu_hash_table_.data(), resolution_, cast_size, device_origins,
      device_dirs, device_end_pts, ignore_unknown, device_max_ranges,
      device_find_end_pts);

  bool* find_end_pts = new bool[cast_size];
  hipMemcpy(find_end_pts, device_find_end_pts, sizeof(bool) * cast_size, hipMemcpyDeviceToHost);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipFree(device_origins);
  hipFree(device_dirs);
  hipFree(device_end_pts);
  hipFree(device_max_ranges);

  return find_end_pts;
}

}  // namespace octomap
