#include "hip/hip_runtime.h"
/*
 * Created on Tue Jun 22 2021
 *
 * Copyright (c) 2021 HITsz-NRSL
 *
 * Author: EpsAvlc
 */

#include "active_mapping/world_representation/cuda_acc/cuda_hashtable.h"
#include <stdio.h>
#include <iostream>

CudaHashTable::~CudaHashTable() {
  hipFree(hash_table_);
}

CUDA_CALLABLE cu_uint64_t hash(cu_uint64_t k) {
  k ^= k >> 16;
  k *= 0x85ebca6b;
  k ^= k >> 13;
  k *= 0xc2b2ae35;
  k ^= k >> 16;
  return k & (kHashTableCapacity - 1);
}

__global__ void gpu_hashtable_insert(KeyValue *hashtable, const KeyValue *kvs,
                                     const unsigned int numkvs) {
  unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadid < numkvs) {
    cu_uint64_t key = kvs[threadid].key;
    cu_uint64_t value = kvs[threadid].value;
    cu_uint64_t slot = hash(key);

    while (true) {
      cu_uint64_t prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
      if (prev == kEmpty || prev == key) {
        hashtable[slot].value = value;
        return;
      } 
      printf("key: %d, slot: %d, value: %d\n", key, slot, value);

      slot = (slot + 1) & (kHashTableCapacity - 1);
    }
  }
}

void CudaHashTable::insert(const KeyValue *kvs, cu_uint64_t num_kvs) {
  // Copy the keyvalues to the GPU
  KeyValue* device_kvs;
  hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
  hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

  // Have CUDA calculate the thread block size
  int mingridsize;
  int threadblocksize;
  hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

  // Create events for GPU timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // Insert all the keys into the hash table
  int gridsize = ((cu_uint64_t)num_kvs + threadblocksize - 1) / threadblocksize;
  gpu_hashtable_insert<<<gridsize, threadblocksize>>>(hash_table_, device_kvs, (cu_uint64_t)num_kvs);

  hipEventRecord(stop);

  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  float seconds = milliseconds / 1000.0f;
  // printf("    GPU inserted %lld items in %f ms (%f million keys/second)\n", 
      // num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

  hipFree(device_kvs);
}

CUDA_CALLABLE cu_uint64_t CudaHashTable::gpuQuery(const KeyValue* hash_table, cu_uint64_t key) {
  cu_uint64_t slot = hash(key);
  while (true) {
    cu_uint64_t prev_key = hash_table[slot].key;
    if (prev_key == kEmpty) {
      return vEmpty;
    } else if (prev_key == key) {
      // printf("query: key: %d, slot: %d, value: %d\n", key, slot, hash_table[slot].value);
      return hash_table[slot].value;
    }

    slot = (slot + 1) & (kHashTableCapacity - 1);
  }
}

__global__ void gpu_hashtable_query(const KeyValue * hash_table, KeyValue *kvs,
                                     const unsigned int numkvs) {
  unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadid < numkvs) {
    cu_uint64_t key = kvs[threadid].key;
    cu_uint64_t slot = hash(key);
    kvs[threadid].value = CudaHashTable::gpuQuery(hash_table, key);
  }
}

void CudaHashTable::query(KeyValue *kvs, cu_uint64_t num_kvs) {
  KeyValue* device_kvs;
  hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
  hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

  // Have CUDA calculate the thread block size
  int mingridsize;
  int threadblocksize;
  hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

  // Create events for GPU timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // Insert all the keys into the hash table
  int gridsize = ((cu_uint64_t)num_kvs + threadblocksize - 1) / threadblocksize;
  gpu_hashtable_query<<<gridsize, threadblocksize>>>(hash_table_, device_kvs, (cu_uint64_t)num_kvs);

  hipEventRecord(stop);

  hipMemcpy(kvs, device_kvs, sizeof(KeyValue) * num_kvs, hipMemcpyDeviceToHost);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  float seconds = milliseconds / 1000.0f;
  printf("    GPU  %lld items in %f ms (%f million keys/second)\n", 
      num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

  hipFree(device_kvs);
}

void CudaHashTable::createHashTable() {
  // Allocate memory
  hipMalloc(&hash_table_, sizeof(KeyValue) * kHashTableCapacity);

  // Initialize hash table to empty
  hipMemset(hash_table_, 0xff, sizeof(KeyValue) * kHashTableCapacity);
}

// void CudaHashT
